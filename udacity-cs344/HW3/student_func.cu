#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Definition Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

[0 1 2 2 3 3]

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"
using namespace std;

float *d_minWorking;

// Reduce to get the min value
__global__
void minReduce(float* const d_values, 
              const size_t numCells)
{
  //
  int x = threadIdx.x;

  int s = 1;
  
  for (int numLeft = numCells; numLeft > 1; s*=2)
  {
    if (x % s == 0 && x + s < numCells)
    {
      d_values[x] = min(d_values[x], d_values[x + s]);
    }
    if (numLeft % 2 == 0) 
    {
      numLeft /= 2;
    }
    else
    {
      numLeft = (numLeft + 1)/2;
    }

    // wait for all threads to finish adding
    __syncthreads();
  }

  // result should be in d_values[0];
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */



  int i = 0;

  cout << "HELP ME\n";

  // calculate the pixel coordinate for this thread
  float value = d_logLuminance[0];

  // allocate second float array of numRows * numCols

  // do a min reduction op
  int curCells = numRows * numCols;

  checkCudaErrors(hipMalloc(&d_minWorking, sizeof(float) * curCells));


  // create array
  float testArray[] = {4,3,26,21,25,3,6,15,10,3,5,6,7};
  int numCells = sizeof(testArray)/sizeof(float);
  cout << "TestArray cells: " << numCells << endl;

  // copy array


  checkCudaErrors(hipMemcpy(d_minWorking, testArray, 
    sizeof(testArray), hipMemcpyHostToDevice));

  // gridSize, blockSize
  minReduce<<<1, numCells>>>(d_minWorking, numCells);


  checkCudaErrors(hipMemcpy(testArray, d_minWorking, 
    sizeof(testArray), hipMemcpyDeviceToHost));

  cout << "Output should be 2\n";
  cout << "Output is " << testArray[0] << endl;

}

